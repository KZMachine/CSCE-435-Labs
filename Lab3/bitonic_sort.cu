/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 64 // 2^9
#define BLOCKS 262144 // 2^15
#define NUM_VALS THREADS*BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %fs\n", elapsed);
}

float random_float()
{
  return (float)rand()/(float)RAND_MAX;
}

void array_print(float *arr, int length) 
{
  int i;
  for (i = 0; i < length; ++i) {
    printf("%1.3f ",  arr[i]);
  }
  printf("\n");
}

void array_fill(float *arr, int length)
{
  srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_float();
  }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(float *values)
{
  float *dev_values;
  size_t size = NUM_VALS * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //MEM COPY FROM HOST TO DEVICE
  // clock_t startTime = clock();
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  // clock_t endTime = clock();
  // printf("Host to device: ");
  // print_elapsed(startTime, endTime);

  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */
  
  int j, k;
  // cudaEvent_t start, stop;
  // cudaEventCreate(&start);
  // cudaEventCreate(&stop);


  clock_t kernalStartTime = clock();
  // cudaEventRecord(start);
  /* Major step */
  hipEventRecord(start);

  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipEventRecord(stop);
  // cudaEventRecord(stop);
  hipDeviceSynchronize();
  clock_t kernalEndTime = clock();
  printf("Kernel time: ");
  print_elapsed(kernalStartTime, kernalEndTime);
  
  //MEM COPY FROM DEVICE TO HOST
  // startTime = clock();
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  // endTime = clock();
  
  // printf("Device to host: ");
  // print_elapsed(startTime, endTime);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float effectiveBW = NUM_VALS*sizeof(float)*2/milliseconds/1e6;
  printf("effective Bandwidth(Gb/s): %f", effectiveBW);
  hipFree(dev_values);
}

int main(void)
{
  printf("Threads: %d, Blocks: %d\n", THREADS, BLOCKS);
  clock_t start, stop;

  float *values = (float*) malloc( NUM_VALS * sizeof(float));
  array_fill(values, NUM_VALS);

  start = clock();
  bitonic_sort(values); /* Inplace */
  stop = clock();

  print_elapsed(start, stop);
}